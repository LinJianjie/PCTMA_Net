#include "hip/hip_runtime.h"
#include "radius_cuda.h"

#include <ATen/cuda/HIPContext.h>

#include "../cluster_utils/utils.cuh"

#define THREADS 1024

template <typename scalar_t>
__global__ void radius_kernel(const scalar_t *x, const scalar_t *y,
                              const int64_t *ptr_x, const int64_t *ptr_y,
                              int64_t *row, int64_t *col, scalar_t radius,
                              int64_t max_num_neighbors, int64_t dim) {

  const int64_t batch_idx = blockIdx.x;

  const int64_t x_start_idx = ptr_x[batch_idx];
  const int64_t x_end_idx = ptr_x[batch_idx + 1];

  const int64_t y_start_idx = ptr_y[batch_idx];
  const int64_t y_end_idx = ptr_y[batch_idx + 1];

  for (int64_t n_y = y_start_idx + threadIdx.x; n_y < y_end_idx;
       n_y += THREADS) {
    int64_t count = 0;
    for (int64_t n_x = x_start_idx; n_x < x_end_idx; n_x++) {
      scalar_t dist = 0;
      for (int64_t d = 0; d < dim; d++) {
        dist += (x[n_x * dim + d] - y[n_y * dim + d]) *
                (x[n_x * dim + d] - y[n_y * dim + d]);
      }
      dist = sqrt(dist);

      if (dist <= radius) {
        row[n_y * max_num_neighbors + count] = n_y;
        col[n_y * max_num_neighbors + count] = n_x;
        count++;
      }

      if (count >= max_num_neighbors) {
        break;
      }
    }
  }
}

torch::Tensor radius_cuda(torch::Tensor x, torch::Tensor y,
                          torch::optional<torch::Tensor> ptr_x,
                          torch::optional<torch::Tensor> ptr_y, double r,
                          int64_t max_num_neighbors) {
  CHECK_CUDA(x);
  CHECK_INPUT(x.dim() == 2);
  CHECK_CUDA(y);
  CHECK_INPUT(y.dim() == 2);
  hipSetDevice(x.get_device());

  if (ptr_x.has_value()) {
    CHECK_CUDA(ptr_x.value());
    CHECK_INPUT(ptr_x.value().dim() == 1);
  } else {
    ptr_x = torch::arange(0, x.size(0) + 1, x.size(0),
                          x.options().dtype(torch::kLong));
  }
  if (ptr_y.has_value()) {
    CHECK_CUDA(ptr_y.value());
    CHECK_INPUT(ptr_y.value().dim() == 1);
  } else {
    ptr_y = torch::arange(0, y.size(0) + 1, y.size(0),
                          y.options().dtype(torch::kLong));
  }
  CHECK_INPUT(ptr_x.value().numel() == ptr_y.value().numel());

  auto row =
      torch::full(y.size(0) * max_num_neighbors, -1, ptr_y.value().options());
  auto col =
      torch::full(y.size(0) * max_num_neighbors, -1, ptr_y.value().options());

  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "radius_kernel", [&] {
    radius_kernel<scalar_t><<<ptr_x.value().size(0) - 1, THREADS, 0, stream>>>(
        x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(),
        ptr_x.value().data_ptr<int64_t>(), ptr_y.value().data_ptr<int64_t>(),
        row.data_ptr<int64_t>(), col.data_ptr<int64_t>(), r, max_num_neighbors,
        x.size(1));
  });

  auto mask = row != -1;
  return torch::stack({row.masked_select(mask), col.masked_select(mask)}, 0);
}
